#include "hip/hip_runtime.h"
#include "math.h"
#include <stdio.h>
#include <vector>
#include "../../src/pyGrid.h"
#include "../../src/pyMol2.h"

#define HB_C12 55332.873 
#define HB_C10 18393.199

// Will interpret the dielectric_model attribute as int because 
// string handling through the kernel may not be straight forward
typedef enum class e_DieletricModel {
    Constant,
    Four_r,
    None
} DieletricModel;

__device__ double distance_squared(double x1, double x2, 
                            double y1, double y2, 
                            double z1, double z2) {
    
    return pow(x2 - x1, 2.0) + pow(y2 - y1, 2.0) + pow(z2 - z1, 2.0); 
}

__device__ double angle(double x1, double x2, double x3,
                    double y1, double y2, double y3,
                    double z1, double z2, double z3 ) {
    
  double ab = sqrt(distance_squared(x1, x2, y1, y2, z1, z2));
  double ac = sqrt(distance_squared(x1, x3, y1, y3, z1, z3));
  double bc = sqrt(distance_squared(x2, x3, y2, y3, z2, z3));

  return 180.0/M_PI * acos((pow(ab, 2.0) + pow(bc, 2.0) - pow(ac, 2.0))/ (2*ab*bc));
}

__global__
void compute_grid_softcore_HB_omp(int npointsx, int npointsy, int npointsz,
                                double grid_spacing, 
                                double xbegin, double ybegin, double zbegin,
                                DieletricModel dielectric_model,
                                double deltaij_es6, double deltaij6,
                                double solvation_alpha, double solvation_beta,
                                double sigma, double diel,
                                int N,
                                int xyz_w, double* xyz, //
                                double* charges,
                                double* radii,
                                double* epsilons_sqrt,
                                int* HBacceptors,
                                int HBdonors_w, int* HBdonors, //
                                double* out_elec_grid,
                                double* out_vdwA_grid,
                                double* out_vdwB_grid,
                                double* out_solv_gauss,
                                double* out_rec_solv_gauss,
                                double* out_hb_donor_grid,
                                double* out_hb_acceptor_grid,
                                int* out_rec_si) {


    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i < npointsx && j < npointsy && k < npointsz)
    {
        double x = i*grid_spacing + xbegin;
        double y = j*grid_spacing + ybegin;
        double z = k*grid_spacing + zbegin;

        double elec = 0.0;
        double vdwA = 0.0;
        double vdwB = 0.0;
        double solv = 0.0;
        double rec_solv = 0.0;
        
        for (int a = 0; a < N; a++) 
        {
            double d2 = distance_squared(x, xyz[a*xyz_w + 0], y, xyz[a*xyz_w + 1], z, xyz[a*xyz_w + 2]);
            double d6 = d2*d2*d2;
            double denom = 0.0;

            if (dielectric_model == Constant)
            {
                denom = pow(d6 + deltaij_es6, 1/3);
                elec += (332.0*charges[a])/(diel*denom);
                solv += ((solvation_alpha * charges[a] * charges[a]) + solvation_beta)
                        * exp((-denom/(2*pow(sigma, 2.0)))) / (pow(sigma, 3.0));
                rec_solv += (4.0/3.0) * M_PI * pow(radii[a], 3.0)
                        * exp((-denom/(2*pow(sigma, 2.0)))) / (pow(sigma, 3.0));
            }
            //TODO: condicional desnecessário?
            else 
            {
                denom = pow(d6 + deltaij_es6, 1/3);
                elec += (332.0*charges[a])/(diel*denom);
                solv += ((solvation_alpha * charges[a] * charges[a]) + solvation_beta)
                        * exp((-denom/(2*pow(sigma, 2.0)))) / (pow(sigma, 3.0));
                rec_solv += (4.0/3.0) * M_PI * pow(radii[a], 3.0)
                        * exp((-denom/(2*pow(sigma, 2.0)))) / (pow(sigma, 3.0));
            }

            denom = (d6 + deltaij6);
            vdwA = (4096*epsilons_sqrt[a] * pow(radii[a], 6.0)) / pow(denom, 2.0);
            vdwB = (128*epsilons_sqrt[a] * pow(radii[a], 3.0)) / denom;
        }

        double hb_donor = 0;

        for (int m = 0; m < sizeof(HBdonors)/sizeof(HBdonors[0]); m++)
        {
            double HB0 = HBdonors[m * HBdonors_w + 0];
            double HB1 = HBdonors[m * HBdonors_w + 1];

            double d2 = distance_squared(xyz[HB1*xyz_w + 0], x, xyz[HB1*xyz_w + 0], y, xyz[HB1*xyz_w + 0], z);
            double d10 = d2*d2*d2*d2*d2;
            double ang = angle(xyz[HB0*xyz_w + 0], xyz[HB0*xyz_w + 1], xyz[HB0*xyz_w + 2],
                      xyz[HB1*xyz_w + 0], xyz[HB1*xyz_w + 1], xyz[HB1*xyz_w + 2], x, y, z);
            double angle_term = (pow(cos(ang * M_PI / 180.0), 4.0));
            hb_donor += (HB_C12/(d10*d2)) - (HB_C10/d10);
        }

        double hb_acceptor = 0;
        {
           double d2 = distance_squared(xyz[HBacceptors[n]*xyz_w + 0], x, xyz[HBacceptors[n]*xyz_w + 1], y, xyz[HBacceptors[n]*xyz_w + 2], z);
           double d10 = pow(d2, 5.0);
           hb_acceptor += (HB_C12/(d10*d2)) - (HB_C10/d10);
        }

        out_elec_grid[(i * npointsx + j) * npointsy + k] = elec;
        out_vdwA_grid[(i * npointsx + j) * npointsy + k] = vdwA;
        out_vdwB_grid[(i * npointsx + j) * npointsy + k] = vdwB;
        out_solv_gauss[(i * npointsx + j) * npointsy + k] = solv;
        out_rec_solv_gauss[(i * npointsx + j) * npointsy + k] = rec_solv;
        out_hb_donor_grid[(i * npointsx + j) * npointsy + k] = hb_donor;
        out_hb_acceptor_grid[(i * npointsx + j) * npointsy + k] = hb_acceptor;
    }

    out_rec_si[0] = 0.0;
    for (int a = 0; a < N ; a++)
    {
        out_rec_si[0] += (solvation_alpha * pow(charges[a], 2.0)) + solvation_beta; 
    }
}

void invoke_compute_grid_softcore_HB_omp(const Grid& grid, const Mol2& rec) {

   double* d_xyz, d_charges, d_radii, d_epsilons_sqrt; 
   int* d_HBacceptors, d_HBdonors;

   double* out_elec_grid, out_vdwA_grid, out_vdwB_grid, out_solv_gauss, out_rec_solv_gauss, out_hb_donor_grid, out_hb_acceptor_grid;
   int* out_rec_si;

   hipMalloc(&d_xyz, rec.xyz.size() * rec.xyz[0].size() * sizeof(double));
   hipMalloc(&d_charges, rec.charges.size() * sizeof(double));
   hipMalloc(&d_radii, rec.radii.size() * sizeof(double));
   hipMalloc(&d_epsilons_sqrt, rec.epsilons_sqrt.size() * sizeof(double));

   hipMalloc(&d_HBdonors, rec.HBdonors.size() * rec.HBdonors[0]*size() * sizeof(int));
   hipMalloc(&d_HBacceptors, rec.HBacceptors.size() * sizeof(int));

   size_t size {(grid.npointsx * grid.npointsy * grid.npointsz) * sizeof(double)};
   hipMalloc(&out_rec_si, size);
   hipMalloc(&out_vdwA_grid, size);
   hipMalloc(&out_vdwB_grid, size);
   hipMalloc(&out_solv_gauss, size);
   hipMalloc(&out_rec_solv_gauss, size);
   hipMalloc(&out_hb_donor_grid, size);
   hipMalloc(&out_hb_acceptor_grid, size);

   hipMalloc(&out_rec_si, 1 * sizeof(int));

   //FIXME: possível memory error
   hipMemcpy(rec.xyz.data(), d_xyz, rec.xyz.size() * rec.xyz[0].size() * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(rec.charges.data(), d_charges, rec.charges.size() * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(rec.radii.data(), d_radii, rec.radii.size() * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(rec.epsilons_sqrt.data(), d_epsilons_sqrt, rec.epsilons_sqrt.size() * sizeof(double, hipMemcpyHostToDevice));

   hipMemcpy(rec.HBacceptors.data(), d_HBacceptors, rec.HBacceptors.size() * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(rec.HBdonors.data(), d_HBdonors, rec.HBdonors.size() * rec.HBdonors[0].size() * sizeof(int), hipMemcpyHostToDevice);

   DieletricModel dieletric_model{};

   switch (grid.Input->dielectric_model)
   {
   case "constant":
      dieletric_model = DieletricModel::Constant;
      break;
   
   default:
      dieletric_model = DieletricModel::None;
   }

   //TODO: ver melhor estes parâmetros de launch
   compute_grid_softcore_HB_omp<<<(size+255)/256, 256>>>(grid.npointsx, grid.npointsy, grid.npointsz,
                                                         grid.grid_spacing,
                                                         grid.xbegin, grid.ybegin, grid.zbegin,
                                                         dieletric_model,
                                                         grid.Input->deltaij_es6, grid.Input->deltaij6,
                                                         grid.Input->solvation_alpha, grid.Input->solvation_beta,
                                                         grid.Input->sigma, grid.Input->diel,
                                                         rec.N,
                                                         rec.xyz[0].size(), d_xyz,//FIXME: pode ser o shape errado 
                                                         d_charges,
                                                         d_radii,
                                                         d_epsilons_sqrt,
                                                         d_HBacceptors,
                                                         rec.HBdonors[0].size(), d_HBdonors,
                                                         out_elec_grid,
                                                         out_vdwA_grid,
                                                         out_vdwB_grid,
                                                         out_solv_gauss,
                                                         out_rec_solv_gauss,
                                                         out_hb_donor_grid,
                                                         out_hb_acceptor_grid,
                                                         out_rec_si)   

   //TODO: verificar como funciona. Se for move semantics, checar se é memory safe
   


   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   hipFree();
   
}

class Bar{
public:
    std::vector<int> val(3, 10);
    Bar() = default;
};

int main() {

    Bar test{};
    Mol2 mol{};
    printf("%d, %d\n", test.val[1], mol.Nbonds);

    return 0;
}
